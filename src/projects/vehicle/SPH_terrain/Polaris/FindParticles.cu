#include "hip/hip_runtime.h"
#include <fstream>

#include <thrust/copy.h>
#include <thrust/gather.h>
#include <thrust/for_each.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/functional.h>
#include <thrust/execution_policy.h>

#include "FindParticles.h"

using namespace chrono;
using namespace chrono::fsi;

struct in_box {
    in_box(const OBBspec& obb) : m_obb(obb) {}

    __device__ bool operator()(const Real4 v) {
        // Convert location in box frame
        auto d = mR3(v) - m_obb.p;
        auto w = mR3(                                                //
            m_obb.ax.x * d.x + m_obb.ax.y * d.y + m_obb.ax.z * d.z,  //
            m_obb.ay.x * d.x + m_obb.ay.y * d.y + m_obb.ay.z * d.z,  //
            m_obb.az.x * d.x + m_obb.az.y * d.y + m_obb.az.z * d.z   //
        );
        // Check w between all box limits
        const Real3& h = m_obb.h;
        return (w.x >= -h.x && w.x <= +h.x) && (w.y >= -h.y && w.y <= +h.y) && (w.z >= -h.z && w.z <= +h.z);
    }

    OBBspec m_obb;
};

struct print_particle_pos {
    print_particle_pos(std::ofstream* stream) : m_stream(stream) {}
    __host__ void operator()(const Real4 p) { (*m_stream) << p.x << ", " << p.y << ", " << p.z << "\n"; }
    std::ofstream* m_stream;
};

struct print_particle_pos_vel {
    print_particle_pos_vel(std::ofstream* stream) : m_stream(stream) {}
    template <typename T>
    __host__ void operator()(const T pv) {
        auto p = thrust::get<0>(pv);
        auto v = thrust::get<1>(pv);
        (*m_stream) << p.x << ", " << p.y << ", " << p.z << ", " << v.x << ", " << v.y << ", " << v.z << "\n";
    }
    std::ofstream* m_stream;
};

thrust::device_vector<int> FindParticlesInBox(std::shared_ptr<ChSystemFsi_impl> sysFSI, const OBBspec& obb) {
    // Extract indices of SPH particles contained in the OBB
    auto ref = sysFSI->fsiGeneralData->referenceArray;
    auto pos_D = sysFSI->sphMarkersD2->posRadD;

    // Find start and end locations for SPH particles (exclude ghost and BCE markers)
    bool haveHelper = (ref[0].z == -3) ? true : false;
    bool haveGhost = (ref[0].z == -2 || ref[1].z == -2) ? true : false;
    auto sph_start = ref[haveHelper + haveGhost].x;
    auto sph_end = ref[haveHelper + haveGhost].y;
    auto num_sph = sph_end - sph_start;

    // Preallocate output vector of indices
    thrust::device_vector<int> indices_D(num_sph);

    // Extract indices of SPH particles inside OBB
    thrust::counting_iterator<int> first(0);
    thrust::counting_iterator<int> last(num_sph);
    auto end = thrust::copy_if(thrust::device,     // execution policy
                               first, last,        // range of all particle indices
                               pos_D.begin(),      // stencil vector
                               indices_D.begin(),  // beginning of destination
                               in_box(obb)         // predicate for stencil elements
    );

    // Trim the output vector of indices
    size_t num_active = (size_t)(end - indices_D.begin());
    indices_D.resize(num_active);

    return indices_D;
}

void WriteParticlePos(std::shared_ptr<ChSystemFsi_impl> sysFSI,
                      const thrust::device_vector<int>& indices_D,
                      const std::string& filename) {
    // Gather positions from particles with specified indices
    const auto& allpos_D = sysFSI->sphMarkersD2->posRadD;

    thrust::device_vector<Real4> pos_D(allpos_D.size());

    auto end = thrust::gather(thrust::device,                      // execution policy
                              indices_D.begin(), indices_D.end(),  // range of gather locations
                              allpos_D.begin(),                    // beginning of source
                              pos_D.begin()                        // beginning of destination
    );

    // Trim the output vector of particle positions
    size_t num_active = (size_t)(end - pos_D.begin());
    assert(num_active == indices_D.size());
    pos_D.resize(num_active);

    // Copy vector to host
    thrust::host_vector<Real4> pos_H = pos_D;

    // Write output file
    std::ofstream stream;
    stream.open(filename, std::ios_base::trunc);
    thrust::for_each(thrust::host, pos_H.begin(), pos_H.end(), print_particle_pos(&stream));
    stream.close();
}

void WriteParticlePosVel(std::shared_ptr<ChSystemFsi_impl> sysFSI,
                         const thrust::device_vector<int>& indices_D,
                         const std::string& filename) {
    // Gather positions and velocities from particles with specified indices
    auto allpos_D = sysFSI->sphMarkersD2->posRadD;
    auto allvel_D = sysFSI->sphMarkersD2->velMasD;

    thrust::device_vector<Real4> pos_D(allpos_D.size());
    thrust::device_vector<Real3> vel_D(allpos_D.size());

    /*
    //// RADU TODO: - this gives errors! Bug in thrust?
    auto end = thrust::gather(
        thrust::device,                                                                     // execution policy
        indices_D.begin(), indices_D.end(),                                                 // range of gather locations
        thrust::make_zip_iterator(thrust::make_tuple(allpos_D.begin(), allvel_D.begin())),  // beginning of source
        thrust::make_zip_iterator(thrust::make_tuple(pos_D.begin(), vel_D.begin()))         // beginning of destination
    );
    */

    auto end = thrust::gather(thrust::device,                      // execution policy
                              indices_D.begin(), indices_D.end(),  // range of gather locations
                              allpos_D.begin(),                    // beginning of source
                              pos_D.begin()                        // beginning of destination
    );

    thrust::gather(thrust::device,                      // execution policy
                   indices_D.begin(), indices_D.end(),  // range of gather locations
                   allvel_D.begin(),                    // beginning of source
                   vel_D.begin()                        // beginning of destination
    );

    // Trim the output vectors of particle positions and velocities
    size_t num_active = (size_t)(end - pos_D.begin());
    assert(num_active == indices_D.size());
    pos_D.resize(num_active);
    vel_D.resize(num_active);

    // Copy vectors to host
    thrust::host_vector<Real4> pos_H = pos_D;
    thrust::host_vector<Real3> vel_H = vel_D;

    // Write output file
    std::ofstream stream;
    stream.open(filename, std::ios_base::trunc);
    thrust::for_each(thrust::host,                                                                 //
                     thrust::make_zip_iterator(thrust::make_tuple(pos_H.begin(), vel_H.begin())),  //
                     thrust::make_zip_iterator(thrust::make_tuple(pos_H.end(), vel_H.end())),      //
                     print_particle_pos_vel(&stream)                                               //
    );
    stream.close();
}
